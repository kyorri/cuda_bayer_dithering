#include "hip/hip_runtime.h"
#include "BayerDithering.h"

#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <algorithm>

namespace bayer_dithering {

    __device__ uint8_t map_dither_device(uint8_t x, uint8_t in_min, uint8_t in_max, uint8_t out_min, uint8_t out_max) {
        return (x - in_min) * (out_max - out_min) / (in_max - in_min) + out_min;
    }

    __global__ void DitheringKernel(Pixel* in_pixels, Pixel* out_pixels, uint32_t width, uint32_t height) {
        uint8_t dither[64] = { 0, 32, 8, 40, 2, 34, 10, 42,
                        48, 16, 56, 24, 50, 18, 58, 26,
                        12, 44, 4, 36, 14, 46, 6, 38,
                        60, 28, 52, 20, 62, 30, 54, 22,
                        3, 35, 11, 43, 1, 33, 9, 41,
                        51, 19, 59, 27, 49, 17, 57, 25,
                        15, 47, 7, 39, 13, 45, 5, 37,
                        63, 31, 55, 23, 61, 29, 53, 21 };
        uint8_t dither_size = 8;

        uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            uint32_t dither_idx = (y % dither_size) * dither_size + (x % dither_size);
            uint8_t dither_value = map_dither_device(dither[dither_idx], 0, dither_size * dither_size - 1, 0, 255);
            float threshold = static_cast<float>(dither_value);

            Pixel p = in_pixels[y * width + x];
            float new_value = 0.0f;

            float accuracy = 1.0f / 256.0f;
            
            if (p.r > threshold - accuracy || p.b > threshold - accuracy || p.g > threshold - accuracy) {
                new_value = 1.0f;
            }

            out_pixels[y * width + x].r = new_value;
            out_pixels[y * width + x].g = new_value;
            out_pixels[y * width + x].b = new_value;
        }
    }



    void BayerDithering::ProcessImageParallel(Image& _img, uint32_t blockX, uint32_t blockY) {
        
        std::vector<Pixel>& img_pixels = _img.GetBuffer();
        uint32_t width = _img.GetWidth();
        uint32_t height = _img.GetHeight();

        Pixel* in_image;
        Pixel* out_image;

        hipMallocManaged(&in_image, width * height * sizeof(Pixel));
        hipMallocManaged(&out_image, width * height * sizeof(Pixel));

        hipMemcpy(in_image, img_pixels.data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);

        dim3 blockDim(blockX, blockY);
        dim3 gridDim((width + blockX - 1) / blockX, (height + blockY - 1) / blockY);

        DitheringKernel<<<gridDim, blockDim >>>(in_image, out_image, width, height);

        hipDeviceSynchronize();
        hipMemcpy(img_pixels.data(), out_image, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

        _img.SetBuffer(img_pixels);

        hipFree(in_image);
        hipFree(out_image);
    }

} // namespace bayer_dithering
