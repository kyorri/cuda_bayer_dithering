#include "hip/hip_runtime.h"
#include "BayerDithering.h"

#include "Image.h"

#include <cmath>
#include <algorithm>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

namespace bayer_dithering {
    __global__ void BayerDitheringKernel(Pixel *img_pixels, uint32_t width, uint32_t height, uint8_t *dither, uint8_t dither_size, uint32_t multiplier) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            uint32_t dither_idx = (y % dither_size) * dither_size + (x % dither_size);
            uint8_t threshold = dither[dither_idx] * multiplier;
            uint32_t flipped_idx = ((height - 1 - y) * width) + x;
            Pixel& p = img_pixels[flipped_idx];
            p = (p.r > threshold || p.g > threshold || p.b > threshold) ? Pixel(255) : Pixel(0);
        }
    }

    void BayerDithering::ProcessImageParallel(Image& _img) {
        _img.ConvertGrayscale();

        std::vector<uint8_t> dither { 0, 32, 8, 40, 2, 34, 10, 42,
                                    48, 16, 56, 24, 50, 18, 58, 26,
                                    12, 44, 4, 36, 14, 46, 6, 38,
                                    60, 28, 52, 20, 62, 30, 54, 22,
                                    3, 35, 11, 43, 1, 33, 9, 41,
                                    51, 19, 59, 27, 49, 17, 57, 25,
                                    15, 47, 7, 39, 13, 45, 5, 37,
                                    63, 31, 55, 23, 61, 29, 53, 21};

        uint8_t dither_size = std::sqrt(dither.size());
        auto max_threshold = std::max_element(dither.begin(), dither.end());
        uint32_t multiplier = 256 / *max_threshold;

        std::vector<Pixel>& img_pixels = _img.GetBuffer();
        uint32_t width = _img.GetWidth();
        uint32_t height = _img.GetHeight();

        Pixel* d_img_pixels;
        uint8_t* d_dither;

        hipMalloc(&d_img_pixels, width * height * sizeof(Pixel));
        hipMalloc(&d_dither, dither.size() * sizeof(uint8_t));

        hipMemcpy(d_img_pixels, img_pixels.data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);
        hipMemcpy(d_dither, dither.data(), dither.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

        BayerDitheringKernel<<<gridDim, blockDim>>>(d_img_pixels, width, height, d_dither, dither_size, multiplier);

        hipMemcpy(img_pixels.data(), d_img_pixels, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

        hipFree(d_img_pixels);
        hipFree(d_dither);
    }

} // namespace bayer_dithering 
